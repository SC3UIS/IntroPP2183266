#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

__host__ __device__ double f(double x){
  return x*x;
}

__device__ double atomicAddDouble(double* address, double val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val + __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}

__global__ void integrate(double a, double b, int n, double h, double* result) {
  int i;
  double x, sum = 0.0;
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  
  for (i = idx + 1; i < n; i += stride) {
    x = a + i * h;
    sum += f(x);
  }
  
  sum *= 2.0;
  
  atomicAddDouble(result, sum);
}

double getCurrentTime() {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return tv.tv_sec + tv.tv_usec * 1e-6;
}

int main(){
  int n, blockSize, numBlocks;
  double a, b, h, integral;
  double* result;
  double* dev_result;
  
  printf("\nEnter the no. of sub-intervals: ");
  scanf("%d", &n);
  printf("\nEnter the initial limit: ");
  scanf("%lf", &a);
  printf("\nEnter the final limit: ");
  scanf("%lf", &b);
  
  h = fabs(b - a) / n;
  
  blockSize = 256; // Puedes ajustar el tamaño del bloque según tus necesidades
  numBlocks = (n + blockSize - 1) / blockSize;
  
  result = (double*)malloc(sizeof(double));
  hipMalloc((void**)&dev_result, sizeof(double));
  
  *result = 0.0;
  hipMemcpy(dev_result, result, sizeof(double), hipMemcpyHostToDevice);
  
  double start = getCurrentTime();
  integrate<<<numBlocks, blockSize>>>(a, b, n, h, dev_result);
  hipDeviceSynchronize();
  double end = getCurrentTime();
  
  hipMemcpy(result, dev_result, sizeof(double), hipMemcpyDeviceToHost);
  
  integral = (h / 2) * (f(a) + f(b) + *result);
  
  printf("\nThe integral is: %lf\n", integral);
  
  double elapsedTime = end - start;
  printf("Elapsed Time: %.6f seconds\n", elapsedTime);
  
  // Cálculo del speedup y la escalabilidad
  int numThreads = numBlocks * blockSize;
  double sequentialTime = integral;
  double parallelTime = elapsedTime;
  double speedup = sequentialTime / parallelTime;
  double scalability = sequentialTime / (parallelTime * numThreads);
  
  printf("Speedup: %.2f\n", speedup);
  printf("Escalabilidad: %.2f\n", scalability);
  
  free(result);
  hipFree(dev_result);
  
  return 0;
}