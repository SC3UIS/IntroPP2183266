#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256

__device__ long long int atomicAddLL(long long int* address, long long int val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, static_cast<unsigned long long int>(val + assumed));
    } while (assumed != old);

    return old;
}

__global__ void sumKernel(int num, long long int* result) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    long long int sum = 0;

    for (int count = idx + 1; count <= num; count += stride) {
        sum += count;
    }

    atomicAddLL(result, sum);
}

int main() {
    int num;
    long long int sum = 0;
    long long int* dev_result;

    printf("Enter a positive integer: ");
    scanf("%d", &num);

    hipMalloc((void**)&dev_result, sizeof(long long int));
    hipMemcpy(dev_result, &sum, sizeof(long long int), hipMemcpyHostToDevice);

    int numBlocks = (num + BLOCK_SIZE - 1) / BLOCK_SIZE;

    sumKernel<<<numBlocks, BLOCK_SIZE>>>(num, dev_result);
    hipDeviceSynchronize();

    hipMemcpy(&sum, dev_result, sizeof(long long int), hipMemcpyDeviceToHost);

    printf("\nSum = %lld\n", sum);

    hipFree(dev_result);

    return 0;
}
