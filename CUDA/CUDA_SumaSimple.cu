#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void sumNumbers(int num, int *result)
{
    int count = threadIdx.x + 1;
    atomicAdd(result, count);
}

double get_wall_time() {
    struct timeval time;
    if (gettimeofday(&time, NULL)) {
        return 0;
    }
    return (double)time.tv_sec + (double)time.tv_usec * 0.000001;
}

int main()
{
    int num, sum = 0;
    int *dev_result;

    printf("Enter a positive integer: ");
    scanf("%d", &num);

    double start_time = get_wall_time(); // Inicio del tiempo de ejecución

    hipMalloc((void**)&dev_result, sizeof(int));
    hipMemcpy(dev_result, &sum, sizeof(int), hipMemcpyHostToDevice);

    sumNumbers<<<1, num>>>(num, dev_result);

    hipMemcpy(&sum, dev_result, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_result);

    double end_time = get_wall_time(); // Fin del tiempo de ejecución
    double tiempo_ejecucion = end_time - start_time;

    printf("\nSum = %d\n", sum);
    printf("Tiempo de ejecución: %f segundos\n", tiempo_ejecucion);

    // Cálculo del speedup y la escalabilidad
    double tiempo_secuencial = num * (num + 1) / 2; // Cálculo secuencial equivalente
    double speedup = tiempo_secuencial / tiempo_ejecucion;
    double escalabilidad = speedup;

    printf("Speedup: %f\n", speedup);
    printf("Escalabilidad: %f\n", escalabilidad);

    return 0;
}