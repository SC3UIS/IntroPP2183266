#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void sumNumbers(int num, int *result)
{
    int count = threadIdx.x + 1;
    atomicAdd(result, count);
}

double get_wall_time()
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return (double)milliseconds / 1000.0;
}

int main()
{
    int num, sum = 0;
    int *dev_result;

    printf("Enter a positive integer: ");
    scanf("%d", &num);

    double start_time = get_wall_time(); // Inicio del tiempo de ejecución

    hipMalloc((void **)&dev_result, sizeof(int));
    hipMemcpy(dev_result, &sum, sizeof(int), hipMemcpyHostToDevice);

    sumNumbers<<<1, num>>>(num, dev_result);

    hipMemcpy(&sum, dev_result, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_result);

    double end_time = get_wall_time(); // Fin del tiempo de ejecución
    double tiempo_ejecucion = end_time - start_time;

    printf("\nSum = %d\n", sum);
    printf("Tiempo de ejecución: %f segundos\n", tiempo_ejecucion);

    // Cálculo de la escalabilidad y el speedup
    double tiempo_secuencial = 0.0;
    double speedup = tiempo_secuencial / tiempo_ejecucion;
    double escalabilidad = speedup;

    printf("Speedup: %f\n", speedup);
    printf("Escalabilidad: %f\n", escalabilidad);

    return 0;
}