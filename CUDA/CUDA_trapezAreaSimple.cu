#include<stdio.h>
#include<math.h>
#include<sys/time.h>
#include<hip/hip_runtime.h>

// Kernel function for parallel computation
__global__ void trapezoidalRule(double a, double b, double h, int n, double* result) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        double x = a + idx * h;
        result[idx] = h * (x * x);
    }
}

double get_wall_time() {
    struct timeval time;
    if (gettimeofday(&time, NULL)) {
        return 0;
    }
    return (double)time.tv_sec + (double)time.tv_usec * 0.000001;
}

int main() {
    int n, i;
    double a, b, h, integral;
    double *result;
    double *d_result;
    struct timeval start, end;

    printf("\nEnter the no. of sub-intervals: ");
    scanf("%d", &n);
    printf("\nEnter the initial limit: ");
    scanf("%lf", &a);
    printf("\nEnter the final limit: ");
    scanf("%lf", &b);

    h = fabs(b - a) / n;

    // Allocate memory for the result array on the host
    result = (double*)malloc(n * sizeof(double));

    // Allocate memory for the result array on the device
    hipMalloc((void**)&d_result, n * sizeof(double));

    double start_time = get_wall_time(); // Inicio del tiempo de ejecución

    // Launch the kernel with one thread per interval
    trapezoidalRule<<<(n + 255) / 256, 256>>>(a, b, h, n, d_result);

    double end_time = get_wall_time(); // Fin del tiempo de ejecución
    double execution_time = end_time - start_time;

    // Copy the result array from the device to the host
    hipMemcpy(result, d_result, n * sizeof(double), hipMemcpyDeviceToHost);

    // Calculate the final integral value
    integral = 0.0;
    for (i = 0; i < n; i++) {
        integral += result[i];
    }
    integral -= (result[0] + result[n - 1]) / 2.0;

    // Free memory on the device
    hipFree(d_result);

    // Free memory on the host
    free(result);

    printf("\nThe integral is: %lf\n", integral);
    printf("Execution Time: %.6f ms\n", execution_time);

    // Cálculo del speedup y la escalabilidad
    double sequential_time = (b * b * b * b - a * a * a * a) / 4; // Cálculo secuencial equivalente
    double speedup = sequential_time / (execution_time / 1000.0);
    double scalability = speedup / 1.0; // Suponiendo que se utiliza 1 GPU

    printf("Speedup: %.2f\n", speedup);
    printf("Scalability: %.2f\n", scalability);

    return 0;
}
