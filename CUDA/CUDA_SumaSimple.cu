#include "hip/hip_runtime.h"
#include <stdio.h>

_global_ void sumNumbers(int num, int *result)
{
    int count = threadIdx.x + 1;
    atomicAdd(result, count);
}

int main()
{
    int num, sum = 0;
    int *dev_result;
    
    printf("Enter a positive integer: ");
    scanf("%d", &num);

    hipMalloc((void**)&dev_result, sizeof(int));
    hipMemcpy(dev_result, &sum, sizeof(int), hipMemcpyHostToDevice);

    sumNumbers<<<1, num>>>(num, dev_result);

    hipMemcpy(&sum, dev_result, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_result);

    printf("\nSum = %d\n", sum);

    return 0;
}