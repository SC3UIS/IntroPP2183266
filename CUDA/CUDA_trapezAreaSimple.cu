
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<sys/time.h>

// Kernel function for parallel computation
__global__ void trapezoidalRule(double a, double b, double h, int n, double* result) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        double x = a + idx * h;
        result[idx] = h * (x * x);
    }
}

// Function to calculate the execution time in milliseconds
double getExecutionTime(struct timeval start, struct timeval end) {
    return (double)(end.tv_sec - start.tv_sec) * 1000.0 +
           (double)(end.tv_usec - start.tv_usec) / 1000.0;
}

int main() {
    int n, i;
    double a, b, h, integral;
    double *result;
    double *d_result;
    struct timeval start, end;

    printf("\nEnter the no. of sub-intervals: ");
    scanf("%d", &n);
    printf("\nEnter the initial limit: ");
    scanf("%lf", &a);
    printf("\nEnter the final limit: ");
    scanf("%lf", &b);

    h = fabs(b - a) / n;

    // Allocate memory for the result array on the host
    result = (double*)malloc(n * sizeof(double));

    // Allocate memory for the result array on the device
    hipMalloc((void**)&d_result, n * sizeof(double));

    // Start timer
    gettimeofday(&start, NULL);

    // Launch the kernel with one thread per interval
    trapezoidalRule<<<(n + 255) / 256, 256>>>(a, b, h, n, d_result);

    // Copy the result array from the device to the host
    hipMemcpy(result, d_result, n * sizeof(double), hipMemcpyDeviceToHost);

    // Calculate the final integral value
    integral = 0.0;
    for (i = 0; i < n; i++) {
        integral += result[i];
    }
    integral -= (result[0] + result[n - 1]) / 2.0;

    // Stop timer
    gettimeofday(&end, NULL);

    // Calculate execution time
    double executionTime = getExecutionTime(start, end);

    // Calculate the speedup and scalability
    double sequentialTime = executionTime / 1000.0; // Assuming sequential execution time in seconds
    double speedup = sequentialTime / (executionTime / 1000.0);
    double scalability = speedup / 1.0; // Assuming 1 GPU used

    // Free memory on the device
    hipFree(d_result);

    // Free memory on the host
    free(result);

    // Print the answer, execution time, speedup, and scalability
    printf("\nThe integral is: %lf\n", integral);
    printf("Execution Time: %.6f ms\n", executionTime);
    printf("Speedup: %.2f\n", speedup);
    printf("Scalability: %.2f\n", scalability);

    return 0;
}