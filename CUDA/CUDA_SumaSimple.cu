#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void sumKernel(int num, int* result) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    int sum = 0;

    for (int count = idx + 1; count <= num; count += stride) {
        sum += count;
    }

    atomicAdd(result, sum);
}

int main() {
    int num, sum = 0;
    int* dev_result;

    printf("Enter a positive integer: ");
    scanf("%d", &num);

    hipMalloc((void**)&dev_result, sizeof(int));
    hipMemcpy(dev_result, &sum, sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;  // Puedes ajustar el tamaño del bloque según tus necesidades
    int numBlocks = (num + blockSize - 1) / blockSize;

    sumKernel<<<numBlocks, blockSize>>>(num, dev_result);
    hipDeviceSynchronize();

    hipMemcpy(&sum, dev_result, sizeof(int), hipMemcpyDeviceToHost);

    printf("\nSum = %d\n", sum);

    hipFree(dev_result);

    return 0;
}
