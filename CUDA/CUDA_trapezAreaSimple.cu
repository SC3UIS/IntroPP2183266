#include<stdio.h>
#include<math.h>
#include<sys/time.h>
#include<hip/hip_runtime.h>

// Kernel function for parallel computation
__global__ void trapezoidalRule(double a, double b, double h, int n, double* result) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        double x = a + idx * h;
        result[idx] = h * (x * x);
    }
}

// Function to calculate the execution time in milliseconds
double getExecutionTime(hipEvent_t start, hipEvent_t end) {
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, end);
    return (double)milliseconds;
}

int main() {
    int n, i;
    double a, b, h, integral;
    double *result;
    double *d_result;
    hipEvent_t start, end;

    printf("\nEnter the no. of sub-intervals: ");
    scanf("%d", &n);
    printf("\nEnter the initial limit: ");
    scanf("%lf", &a);
    printf("\nEnter the final limit: ");
    scanf("%lf", &b);

    h = fabs(b - a) / n;

    // Allocate memory for the result array on the host
    result = (double*)malloc(n * sizeof(double));

    // Allocate memory for the result array on the device
    hipMalloc((void**)&d_result, n * sizeof(double));

    // Start timer
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);

    // Launch the kernel with one thread per interval
    trapezoidalRule<<<(n + 255) / 256, 256>>>(a, b, h, n, d_result);

    // Copy the result array from the device to the host
    hipMemcpy(result, d_result, n * sizeof(double), hipMemcpyDeviceToHost);

    // Calculate the final integral value
    integral = 0.0;
    for (i = 0; i < n; i++) {
        integral += result[i];
    }
    integral -= (result[0] + result[n - 1]) / 2.0;

    // Stop timer
    hipEventRecord(end);
    hipEventSynchronize(end);

    // Calculate execution time
    double executionTime = getExecutionTime(start, end);

    // Calculate the speedup and scalability
    double sequentialTime = executionTime / 1000.0; // Assuming sequential execution time in seconds
    double speedup = sequentialTime / (executionTime / 1000.0);
    double scalability = speedup / 1.0; // Assuming 1 GPU used

    // Free memory on the device
    hipFree(d_result);

    // Free memory on the host
    free(result);

    // Print the answer, execution time, speedup, and scalability
    printf("\nThe integral is: %lf\n", integral);
    printf("Execution Time: %.6f ms\n", executionTime);
    printf("Speedup: %.2f\n", speedup);
    printf("Scalability: %.2f\n", scalability);

    return 0;
}
